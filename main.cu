#include "hip/hip_runtime.h"
#include <aicuda_stl_map.h>
#include <aicuda_stl_vector.h>
#include <aicuda_stl_string.h>
#include <aicuda_stl_list.h>
#include <aicuda_stl_set.h>
#include <stdio.h>

__global__ void
vectorAdd()
{
    printf("vectorAdd()\n");
    aicuda::stl::map<int, int> mp1;
    mp1[1] = 1;
    mp1[2] = 2;
    mp1[3] = 3;
    for(aicuda::stl::map<int, int>::const_iterator itr = mp1.begin();
        itr != mp1.end(); ++itr)
    {
        printf("kernel [ %d, %d ]\n", itr->first, itr->second);
    }

    aicuda::stl::vector<int> vt;
    aicuda::stl::string a("111");
    a = "222";
    printf("a = %s \n", a.c_str());

    vt.push_back(111);
    vt.push_back(222);

    aicuda::stl::map<int, aicuda::stl::vector<int> > mp2;
    mp2[1].push_back(1);
    mp2[1].push_back(11);
    mp2[1].push_back(111);
    mp2[2].push_back(2);
    mp2[2].push_back(22);
    mp2[2].push_back(222);
    for(aicuda::stl::map<int, aicuda::stl::vector<int> >::const_iterator itr = mp2.begin();
        itr != mp2.end(); ++itr)
    {
        printf("kernel [ %d = ", itr->first);
        size_t vt_count = itr->second.size();
        if(vt_count > 0)
        {
            printf("%d",itr->second[0]);
            for(size_t i = 1; i < vt_count; ++i)
            {
                printf(", %d",itr->second[i]); 
            }
            printf(" ]\n");
        }
        else
        {
            printf(" null ]\n");
        }
    }

    aicuda::stl::set<int> st;
    st.insert(1);
    st.insert(2);
    st.insert(1);
    st.insert(2);
    printf("st size = %ld \n", st.size());

    aicuda::stl::list<int> lt;
    lt.push_back(1);
    lt.push_back(2);
    lt.push_back(3);
    lt.push_front(5);

    lt.remove(3);

    for(aicuda::stl::list<int>::const_iterator itr = lt.begin();
        itr != lt.end(); ++itr)
    {
        printf("lt = %d\n", *itr);
    }

    printf("to_string(123) = %s\n", aicuda::stl::string::to_string(123).c_str());
    printf("to_string(\"123\") = %s\n", aicuda::stl::string::to_string("123").c_str());
    //printf("to_string(\"lt\") = %s\n", aicuda::stl::string::to_string(lt).c_str());
    printf("to_string(123.0) = %s\n", aicuda::stl::string::to_string(123.0).c_str());
    printf("to_string(123.0001) = %s\n", aicuda::stl::string::to_string(123.0001).c_str());
    printf("to_string(false) = %s\n", aicuda::stl::string::to_string(false).c_str());
    printf("to_string(true) = %s\n", aicuda::stl::string::to_string(true).c_str());


}

int main()
{
    printf("hello word\n");

    hipError_t cudaStatus = hipDeviceSetLimit(hipLimitStackSize, 200 * 1024);
    if (cudaStatus != hipSuccess) {
        printf("hipDeviceSetLimit returned error after launching Kernel : %s!\n", hipGetErrorString(cudaStatus));
        return -1;
    }

    vectorAdd<<<1, 1>>>();
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        printf("hipDeviceSynchronize returned error after launching Kernel : %s!\n", hipGetErrorString(cudaStatus));
    }

    hipDeviceReset();
}